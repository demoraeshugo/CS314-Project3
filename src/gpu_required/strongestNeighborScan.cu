/*
 **********************************************
 *  CS314 Principles of Programming Languages *
 *  Spring 2020                               *
 **********************************************
 */

 //	       ./gsn ./testcases/input5.mtx output.txt 200


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void strongestNeighborScan_gpu(int * src, int * oldDst, int * newDst, int * oldWeight, int * newWeight, int * madeChanges, int distance, int numEdges) {
	/*YOUR CODE HERE*/
	/* The graph is encoded as an edge list consisting of three arrays: src, dst, and weight, such
	that src[n] is the source node for the n-th edge, dst[n] is the destination node for the n-th
	edge, and weight[n] is the weight of the n-th edge. The graph is undirected, so if src[n]=x
	and dst[n]=y then there exists an edge m such that src[m]=y and dst[m]=x. */

	//Get total num of threads
	int numThreads = blockDim.x * gridDim.x;

	//Get thread ID 
	int tID = blockIdx.x * blockDim.x + threadIdx.x;

	//Case where more threads than needed
	if(tID >= numEdges) return;

	for(int i = tID; i < numEdges; i += numThreads) {
		//printf("tID : %d of %d -------------- Doing work on src[%d]\n", tID, numThreads, i);
		//Current node
		int rightIndex = i;

		//Stride away node
		int leftIndex = i - distance;

		//Enforce array bound
		if(leftIndex < 0) { leftIndex = 0; };

		//Only compare nodes in the same segment
		if(src[leftIndex] == src[rightIndex]) {
			int strongerIndex;
			
			//Get stronger node
			if(oldWeight[leftIndex] > oldWeight[rightIndex]) { 
				strongerIndex = leftIndex; 
			} else if(oldWeight[leftIndex] < oldWeight[rightIndex]){ 
				strongerIndex = rightIndex; 
			} else {
				//if equal weights, take node with smaller vID
				if(oldDst[leftIndex] < oldDst[rightIndex]) { 
					strongerIndex = leftIndex; 
				} else { 
					strongerIndex = rightIndex; 
				};
			}

			//Set new destination
			newDst[i] = oldDst[strongerIndex];

			//Set new weight
			newWeight[i] = oldWeight[strongerIndex];

			//Flag any changes
			if((newDst[i] != oldDst[i]) || (newWeight[i] != oldWeight[i])) { *madeChanges = 1; };

		} else {
			//Different segments defaults to no change
			newDst[i] = oldDst[i];
			newWeight[i] = oldWeight[i];
		}
	}
}

/*
 * Performs segment scan to find strongest neighbor for each src node
 * @param src The source array in the edge list
 * @param oldDst The current dst array in the edge list
 * @param newDst The modified dst array produced by this GPU kernel function
 * @param oldWeight The current weight array in the edge list
 * @param newWeight The modified weight array produced by this GPU kernel function
 * @param madeChanges If our output is different than our input then we must set *madeChanges to 1, so the host will know to launch another step of the scan.
 * @param distance The distance between array locations being examined. This is always a power of 2.
 * @param numEdges The size of the index, weight, and flags arrays.
*/